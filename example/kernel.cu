#include "hip/hip_runtime.h"
//
//  kernel.cu
//  example
//
//  Created by peta on 2013/05/23.
//  Copyright (c) 2013年 peta.okechan.net. All rights reserved.
//

texture<float, hipTextureType1D, hipReadModeElementType> twos;
__device__ float ones[10];

extern "C"
{
    __global__ void addone(float *v, int n)
    {
        int index = blockDim.x * blockIdx.x + threadIdx.x;

        if (index < n) {
            v[index] += tex1D(twos, float(index) / 9.9999f) - ones[index % 10];
        }
    }
}
