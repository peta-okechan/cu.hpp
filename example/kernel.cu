
#include <hip/hip_runtime.h>
//
//  kernel.cu
//  example
//
//  Created by peta on 2013/05/23.
//  Copyright (c) 2013年 peta.okechan.net. All rights reserved.
//

__device__ float ones[10];

extern "C"
{
    __global__ void addone(float *v, int n)
    {
        int index = blockDim.x * blockIdx.x + threadIdx.x;

        if (index < n) {
            v[index] += ones[index % 10];
        }
    }
}
